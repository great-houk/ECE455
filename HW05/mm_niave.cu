#include "hip/hip_runtime.h"
template <typename T>
__global__ void mm_kernel(T const* mat_1,
						  T const* mat_2,
						  T* mat_3,
						  size_t m,
						  size_t n,
						  size_t p) {
	// Compute (i, j) coordinates from 2D grid
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;

	// Boundary check
	if ((i >= m) || (j >= p))
		return;

	// Compute dot product of row i (A) and column j (B)
	T acc_sum = 0;
	for (size_t k = 0; k < n; ++k)
		acc_sum += mat_1[i * n + k] * mat_2[k * p + j];

	mat_3[i * p + j] = acc_sum;	 // Write result
}

#include "helpers.cu"

#include "main.cu"